/*
 * Kobe Davis
 * Prof. Karavan
 * CS 405
 * 19 April 2019
 *
 * Assignment 1: Vector Addition
*/


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__host__   void errCatch(hipError_t err);
__global__ void vecAddKernel(int* A, int* B, int* C, int size);

int main()
{
	int len = 1024;
	int* d_A, * d_B, * d_C;
	int* h_A = new int[len];
	int* h_B = new int[len];
	int* h_C = new int[len];

	for (int i = 0; i < len; ++i) {
		h_A[i] = h_B[i] = 1;
		h_C[i] = 0;
	}

	int size = len * sizeof(int);
	errCatch(hipMalloc((void**)& d_A, size));
	errCatch(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
	errCatch(hipMalloc((void**)& d_B, size));
	errCatch(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
	errCatch(hipMalloc((void**)& d_C, size));

	vecAddKernel<<< len / 256, 256 >>>(d_A, d_B, d_C, len);
	errCatch(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

	int sum = 0;
	cout << "Sum of Resultant Vector: ";
	for (int i = 0; i < len; ++i)
		sum += h_C[i];
	cout << sum << endl;

	errCatch(hipFree(d_A));
	errCatch(hipFree(d_B));
	errCatch(hipFree(d_C));
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;

	return 0;
}

__global__
void vecAddKernel(int* A, int* B, int* C, int size) {
	int idx = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (idx < size)
		C[idx] = A[idx] + B[idx];
}

void errCatch(hipError_t err) {
	if (err != hipSuccess) {
		cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
		exit(EXIT_FAILURE);
	}
}